#include "hip/hip_runtime.h"
#include "crypt_mrm.cuh"

/*
* Seed that generate the current crypto scheme.
*/
__device__ long crypt_mrm_rnd_seed;

/*
* Function that generate a random 0<int<256 value.
*/
__device__ int crypt_mrm_generate_random_byte() {
	crypt_mrm_rnd_seed = (crypt_mrm_rnd_seed * 0x5DEECE66DL + 0xBL) & ((1L << 48) - 1);
	return (unsigned char)(crypt_mrm_rnd_seed >> 16)  % 256;
}

/*
* Current session key.
*/
__device__ unsigned char * crypt_mrm_session_key;
__device__ unsigned char * crypt_mrm_session_table;

/*
* Starts a new crypt mrm session.
*/
void crypt_mrm_init() {
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	crypt_mrm_generate_session << <1, 1 >> >(time(NULL));

	hipDeviceSynchronize();
}

/*
* Generate a new key based in a given seed.
*/
__global__ void crypt_mrm_generate_session(long seed) {
	int i, temp, j;

	crypt_mrm_rnd_seed = seed;
	crypt_mrm_session_key = (unsigned char*)malloc(sizeof(char) * 256);
	crypt_mrm_session_table = (unsigned char*)malloc(sizeof(char) * 256);

	for (i = 0; i < 256; i++)
		crypt_mrm_session_key[i] = crypt_mrm_generate_random_byte();
	
	for (i = 0; i < 256; i++)
		crypt_mrm_session_table[i] = i;

	for (i = j= 0; i < 256; i++) {
		j = (j + crypt_mrm_session_key[i] + crypt_mrm_session_table[i]) & 255;

		temp = crypt_mrm_session_table[i];
		crypt_mrm_session_table[i] = crypt_mrm_session_table[j];
		crypt_mrm_session_table[j] = temp;
	}
}

/*
* The magic starts here.
*/
__global__ void crypt_mrm_magic_bytes(char * dst, const char * src, int size) {
	int i, j, k, temp;
	char *table;

	table = (char*)malloc(sizeof(char) * 256);

	for (i = 0; i < 256; i++)
		table[i] = crypt_mrm_session_table[i];

	for (i = j = k = 0; k < size; k++) {
		i = ++i & 255;
		j = (j + table[i]) & 255;

		temp = table[i];
		table[i] = table[j];
		table[j] = temp;

		dst[k] = src[k] ^ (table[(table[i] + table[j]) & 255]);
	}
	
	free(table);

}

void crypt_mrm_ram_swap(const char * value, char * dst_cpu, size_t size) {
	char * src, *dst;

	hipMalloc((void**)&src, size);
	hipMalloc((void**)&dst, size);

	hipMemcpy(src, value, size, hipMemcpyHostToDevice);

	crypt_mrm_magic_bytes << <1, 1 >> >(dst, src, size);
	hipDeviceSynchronize();

	hipMemcpy(dst_cpu, dst, size, hipMemcpyDeviceToHost);

	hipFree(src);
	hipFree(dst);

}
void crypt_mrm_set_ram_swap_int(int * ptr, int value) {
	*ptr = crypt_mrm_get_ram_swap_int(value);
}

int crypt_mrm_get_ram_swap_int(int value) {
	char * dst_cpu;
	dst_cpu = (char*)malloc(sizeof(int));

	crypt_mrm_ram_swap((char*)(&value), dst_cpu, sizeof(int));
	return *(int *)dst_cpu;
}

void crypt_mrm_set_ram_swap_long(long * ptr, long value) {
	*ptr = crypt_mrm_get_ram_swap_long(value);
}

long crypt_mrm_get_ram_swap_long(long value) {
	char * dst_cpu;
	dst_cpu = (char*)malloc(sizeof(long));

	crypt_mrm_ram_swap((char*)(&value), dst_cpu, sizeof(long));
	return *(long *)dst_cpu;
}
